/*
* This program uses the host CURAND API to generate 10 pseudorandom floats.
*/

#include <stdio.h> 
#include <stdlib.h>
#include <hip/hip_runtime.h> 
#include <hiprand/hiprand.h>

int main(int argc, char *argv[]){
  size_t n = 10;
  size_t i; 
  hiprandGenerator_t gen; 
  float *devData , *hostData;
  
  /* Allocate n floats on host */
  hostData = (float *) calloc(n, sizeof(float));
  
  /* Allocate n floats on device */ 
  hipMalloc((void **) &devData, n*sizeof(float));

  /* Create a Mersenne Twister pseudorandom number generator */ 
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MTGP32);

  /* Set seed */ 
  hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);
  
  /* Generate n floats on device */
  hiprandGenerateUniform(gen, devData, n);
  
  /* Copy device memory to host */ 
  hipMemcpy(hostData , devData , n * sizeof(float), hipMemcpyDeviceToHost);

  /* Show result */
  printf("Random Unif(0, 1) draws:\n");
  for(i = 0; i < n; i++) {
    printf("  %1.4f\n", hostData[i]); 
  }
  printf("\n");

  /* Cleanup */ 
  hiprandDestroyGenerator(gen); 
  hipFree(devData);
  free(hostData);
}