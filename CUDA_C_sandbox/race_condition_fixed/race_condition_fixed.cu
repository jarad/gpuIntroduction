#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
 

__global__ void colonel(int *a_d){
  *a_d += 1;
}

int main(){

  int a = 0, *a_d;
  
  hipMalloc((void**) &a_d, sizeof(int));
  hipMemcpy(a_d, &a, sizeof(int), hipMemcpyHostToDevice);

  float   elapsedTime;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord( start, 0 );

  colonel<<<1000,1000>>>(a_d); 
  
  hipEventRecord( stop, 0 );
  hipEventSynchronize( stop );
  hipEventElapsedTime( &elapsedTime, start, stop );
  hipEventDestroy( start );
  hipEventDestroy( stop );
  printf("GPU Time elapsed: %f seconds\n", elapsedTime/1000.0);
  
  
  hipMemcpy(&a, a_d, sizeof(int), hipMemcpyDeviceToHost);

  printf("a = %d\n", a);
  hipFree(a_d);

}