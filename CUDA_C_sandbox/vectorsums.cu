#define N 10

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void add(int *a, int *b, int *c){
  int tid = blockIdx.x;
  if(tid < N)
    c[tid] = a[tid] + b[tid];
}


int main(void) {
  int i, a[N], b[N], c[N];
  int *dev_a, *dev_b, *dev_c;

  hipMalloc((void**) &dev_a, N*sizeof(int));
  hipMalloc((void**) &dev_b, N*sizeof(int));
  hipMalloc((void**) &dev_c, N*sizeof(int));

  for(i=0; i<N; i++){
    a[i] = -i;
    b[i] = i*i;
  }

  hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);

  add<<<N,1>>>(dev_a, dev_b, dev_c);

  hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);

  for(i = 0; i<N; i++){
    printf("%d + %d = %d \n", a[i], b[i], c[i]);
  }

  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  return 0;
}