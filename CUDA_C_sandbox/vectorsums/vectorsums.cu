#define N 1000000000

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void add(int *a, int *b, int *c){
  int tid = blockIdx.x;
  if(tid < N)
    c[tid] = a[tid] + b[tid];
}


int main(void) {
  int i, a[N], b[N], c[N];
  int *dev_a, *dev_b, *dev_c;

  hipMalloc((void**) &dev_a, N*sizeof(int));
  hipMalloc((void**) &dev_b, N*sizeof(int));
  hipMalloc((void**) &dev_c, N*sizeof(int));

  for(i=0; i<N; i++){
    a[i] = -i;
    b[i] = i*i;
  }

  hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);

  printf("Adding...");
  add<<<N,1>>>(dev_a, dev_b, dev_c);
  printf("Done.\n");

  printf("Clearing memory...");
  hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);

  // printf("\ni =  : \t a[i] \t + \t b[i] \t = \t c[i] \n \n");
  // for(i = 0; i<N; i++){
  //   printf("i = %i: \t %d \t + \t %d \t = \t %d \n", i, a[i], b[i], c[i]);
  // }

  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);
  printf("Done.\n");

  return 0;
}