#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
 

#define N 10

__global__ void add(int *a, int *b, int *c){
  int bid = blockIdx.x;
  if(bid < N)
    c[bid] = a[bid] + b[bid];
}

int main(void) {
  int i, a[N], b[N], c[N];
  int *dev_a, *dev_b, *dev_c;

  hipMalloc((void**) &dev_a, N*sizeof(int));
  hipMalloc((void**) &dev_b, N*sizeof(int));
  hipMalloc((void**) &dev_c, N*sizeof(int));

  for(i=0; i<N; i++){
    a[i] = -i;
    b[i] = i*i;
  }

  hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);

  add<<<N,1>>>(dev_a, dev_b, dev_c);

  hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);

  printf("\na + b = c\n");
  for(i = 0; i<N; i++){
    printf("%5d + %5d = %5d\n", a[i], b[i], c[i]);
  }

  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);
}