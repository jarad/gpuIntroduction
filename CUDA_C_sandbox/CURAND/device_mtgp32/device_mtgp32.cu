#include "hip/hip_runtime.h"
/*
 * This program uses the device CURAND API to calculate what 
 * proportion of pseudo-random ints are odd.
 */

#include <stdio.h>
#include <stdlib.h> 
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

__global__ void setup_kernel(hiprandState *state){
  int id = threadIdx.x + blockIdx.x * 64;

  /* Each thread gets same seed, a different sequence number , no offset */
  hiprand_init(1234, id, 0, &state[id]); 
}
  
__global__ void generate_kernel(hiprandState *state, int *result){
  int id = threadIdx.x + blockIdx.x * 64; int count = 0;
  unsigned int x;

  /* Copy state to local memory for efficiency */ 
  hiprandState localState = state[id];
  
  /* Generate pseudo -random unsigned ints */ 
  for(int n = 0; n < 100000; n++){
    x = hiprand(&localState); 
    
    /* Check if odd */ 
    if(x & 1){
      count ++; 
    }
  }

  /* Copy state back to global memory */ 
  state[id] = localState;

  /* Store results */
  result[id] += count;
}

int main(int argc, char *argv[]){
  int i, total;

 int *devResults, *hostResults;
  hiprandState *devStates;

  /* Allocate space for results on host */ 
  hostResults = (int *) calloc(64 * 64, sizeof(int));
  
  /* Allocate space for results on device */ 
  hipMalloc((void **)&devResults , 64 * 64 *sizeof(int));
  
  /* Set results to 0 */ 
  hipMemset(devResults , 0, 64 * 64 * sizeof(int));
  
  /* Allocate space for prng states on device */ 
  hipMalloc((void **)&devStates , 64 * 64 * sizeof(hiprandState)); 
  
  /* Setup prng states */
 // setup_kernel<<<64, 64>>>(devStates);
  
  hiprandStateMtgp32_t *s;
  mtgp32_params_fast_t params[200];
  mtgp32_kernel_params_t *p;
  int n = 64 * 64;
  unsigned long long seed = 1234;
  
  hiprandMakeMTGP32Constants(params, p);
  hiprandMakeMTGP32KernelState(s, params, *p, n, seed);
  
  
  /* Generate and use pseudorandom numbers*/ 
  for(i = 0; i < 10; i++){
    generate_kernel<<<64, 64>>>(devStates, devResults);
  }
  
  /* Copy device memory to host */ 
  hipMemcpy(hostResults, devResults , 64 * 64 * sizeof(int), hipMemcpyDeviceToHost);

  /* Show result */
  total = 0;
  for(i = 0; i < 64 * 64; i++) {
    total += hostResults[i];
  }
  printf("Fraction odd was %10.13f\n", (float) total / (64.0f * 64.0f * 100000.0f * 10.0f)); 
  
  /* Cleanup */
  hipFree(devStates);
  hipFree(devResults);
  free(hostResults);
  
  return EXIT_SUCCESS;
}