#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
 

__global__ void colonel(int *a_d){
  *a_d = blockDim.x * blockIdx.x + threadIdx.x;
}

int main(){

  int a = 0, *a_d;
  
  hipMalloc((void**) &a_d, sizeof(int));
  hipMemcpy(a_d, &a, sizeof(int), hipMemcpyHostToDevice);

  colonel<<<4,5>>>(a_d); 
  
  hipMemcpy(&a, a_d, sizeof(int), hipMemcpyDeviceToHost);

  printf("a = %d\n", a);
  hipFree(a_d);

}