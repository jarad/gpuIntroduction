#include "hip/hip_runtime.h"
#include "../common/book.h" 
#include "../common/lock.h"

#define imin(a,b) (a<b?a:b)

// NOTE: COMPILE LIKE THIS:
//nvcc dot_product_atomic.cu -arch sm_20 -o dot_product_atomic

const int N = 32 * 1024 * 1024; 
const int threadsPerBlock = 256; 
const int blocksPerGrid =
  imin( 32, (N+threadsPerBlock-1) / threadsPerBlock );
  
__global__ void dot( Lock lock, float *a, 
                     float *b, float *c ) {
                     
  __shared__ float cache[threadsPerBlock];
  int tid = threadIdx.x + blockIdx.x * blockDim.x; 
  int cacheIndex = threadIdx.x;
  
  float temp = 0; 
  while (tid < N) {
        temp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
  }
    
  // set the cache values
  cache[cacheIndex] = temp;

  // synchronize threads in this block
  __syncthreads();

  // for reductions, threadsPerBlock must be a power of 2 
  // because of the following code
  int i = blockDim.x/2;
  while (i != 0) {
    if (cacheIndex < i)
      cache[cacheIndex] += cache[cacheIndex + i];
    __syncthreads();
    i /= 2; 
  }
  
  // Here's where locks come in:
  if (cacheIndex == 0) { lock.lock();
        *c += cache[0];
        lock.unlock();
  }
}

int main( void ) {
  float *a, *b, c = 0;
  float *dev_a, *dev_b, *dev_c;

  // allocate memory on the CPU side
  a = (float*)malloc( N*sizeof(float) ); 
  b = (float*)malloc( N*sizeof(float) );
  
  // allocate the memory on the GPU
  HANDLE_ERROR( hipMalloc( (void**)&dev_a, 
                            N*sizeof(float) ) );
  HANDLE_ERROR( hipMalloc( (void**)&dev_b, 
                            N*sizeof(float) ) );
  HANDLE_ERROR( hipMalloc( (void**)&dev_c, 
                            sizeof(float) ) );

  // fill in the host memory with data
  for (int i=0; i<N; i++) { 
    a[i] = i;
    b[i] = i*2; 
  }
  
  // copy the arrays 'a' and 'b' to the GPU
  HANDLE_ERROR( hipMemcpy( dev_a, a, N*sizeof(float), 
                            hipMemcpyHostToDevice ) );
  HANDLE_ERROR( hipMemcpy( dev_b, b, N*sizeof(float), 
                            hipMemcpyHostToDevice ) );
  HANDLE_ERROR( hipMemcpy( dev_c, &c, sizeof(float), 
                            hipMemcpyHostToDevice ) );

  Lock lock;
  dot<<<blocksPerGrid,threadsPerBlock>>>( lock, dev_a,
                                          dev_b, dev_c );
  
  // copy c back from the GPU to the CPU
  HANDLE_ERROR( hipMemcpy( &c, dev_c, 
                            sizeof(float),
                            hipMemcpyDeviceToHost ) );
                                                      
  #define sum_squares(x) (x*(x+1)*(2*x+1)/6) 
  printf( "Does GPU value %.6g = %.6g?\n", c,
           2 * sum_squares( (float)(N - 1) ) );
           
  // free memory on the GPU side
  hipFree( dev_a );
  hipFree( dev_b );
  hipFree( dev_c );

  // free memory on the CPU side
  free( a );
  free( b ); 
}
                            
                            