#include "hip/hip_runtime.h"
#include <stdio.h> 
#include <stdlib.h> 
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h> 

__global__ void some_kernel(...){...}

int main (void){ 
  // Declare all variables.
  ...
  // Dynamically allocate host memory.
  ...
  // Dynamically allocate device memory.
  ...
  // Write to host memory.
  ... 
  // Copy host memory to device memory.
  ...
  // Execute kernel on the device.
  some_kernel<<< num_blocks, num_theads_per_block >>>(...);
  
  // Write device memory back to host memory.
  ...
  // Free dynamically-allocated host memory
  ...
  // Free dynamically-allocated device memory    
  ...
}