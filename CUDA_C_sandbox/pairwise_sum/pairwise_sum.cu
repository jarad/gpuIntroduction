#include <stdio.h> 
#include <stdlib.h> 
#include <math.h>
#include <hip/hip_runtime.h>
 

/*
 * This program computes the sum of the elements of 
 * vector v using the pairwise (cascading) sum algorithm.
 */

#define N 64 // length of vector v. MUST BE A POWER OF 2.

// Fill the vector v with n random floating point numbers.
void vfill(float* v, int n){
  int i;
  for(i = 0; i < n; i++){
    v[i] = (float) rand() / RAND_MAX;
  }
}

// Print the vector v.
void vprint(float* v, int n){
  int i;
  printf("v = \n");
  for(i = 0; i < n; i++){
    printf("%7.3f\n", v[i]);
  }
  printf("\n");
}

// Pairwise-sum the elements of vector v and store the result in v[0]. 
__global__ void psum(float* v){ 
  int t = threadIdx.x; // Thread index.
  int n = blockDim.x; // Should be half the length of v.

  while (n != 0) {
    if(t < n)
      v[t] += v[t + n];  
    __syncthreads();    
    n /= 2; 
  }
}

int main (void){ 
  float *v_h, *v_d; // host and device copies of our vector, respectively
  
  // dynamically allocate memory on the host for v_h
  v_h = (float*) malloc(N * sizeof(*v_h)); 
  
  // dynamically allocate memory on the device for v_d
  hipMalloc ((float**) &v_d, N *sizeof(*v_d)); 
  
  // Fill v_h with N random floating point numbers.
  vfill(v_h, N);
  
  // Print v_h to the console
  vprint(v_h, N);
  
  // Write the contents of v_h to v_d
  hipMemcpy( v_d, v_h, N * sizeof(float), hipMemcpyHostToDevice );
  
  // Compute the pairwise sum of the elements of v_d and store the result in v_d[0].
  psum<<< 1, N/2 >>>(v_d);
  
  // Write the pairwise sum, v_d[0], to v_h[0].
  hipMemcpy(v_h, v_d, sizeof(float), hipMemcpyDeviceToHost );
  
  // Print the pairwise sum.
  printf("Pairwise sum = %7.3f\n", v_h[0]);
  
  // Free dynamically-allocated host memory
  free(v_h);

  // Free dynamically-allocated device memory    
  hipFree(&v_d);
}