#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <cula.h>
#include <math.h>

#define I(i, j, ld) j * ld + i

float rnorm(){
  float r1 = ((float) rand()) / ((float) RAND_MAX);
  float r2 = ((float) rand()) / ((float) RAND_MAX);
  return sqrt( -2 * log(r1) ) * cos(2 * 3.1415 * r2);
}

int main(){
  int i, j;
  int n = 10;
  int p = 3;
  int* ipiv;
  float k;
  float *X, *XtX, *XtY, *beta, *Y, *dX, *dXtX, *dXtY, *dbeta, *dY;
  
  float *a, *b;
  a = (float*) malloc(sizeof(*X));
  b = (float*) malloc(sizeof(*X));
  *a = 1.0;
  *b = 0.0;
  
  hipblasHandle_t handle;
  hipblasCreate(&handle);
   
  X = (float*) malloc(n * p * sizeof(*X));
  XtX = (float*) malloc(p * p * sizeof(*X));
  XtY = (float*) malloc(p * sizeof(*X));
  beta = (float*) malloc(p * sizeof(*X));
  Y = (float*) malloc(n * sizeof(*X));
  
  hipMalloc((void**) &ipiv, p * p * sizeof(*ipiv));
  hipMalloc((void**) &dX, n * p * sizeof(*X));
  hipMalloc((void**) &dXtX, p * p * sizeof(*X));
  hipMalloc((void**) &dXtY, p * sizeof(*X));
  hipMalloc((void**) &dbeta, p * sizeof(*X));
  hipMalloc((void**) &dY, n * sizeof(*X));

  printf("Y\t\tX\n");
  for(i = 0; i < n; i++){
    k = (float) i;
    X[I(i, 0, n)] = 1.0;
    X[I(i, 1, n)] = k / 10.0;
    X[I(i, 2, n)] = k * k / 10.0;  
    Y[i] = (k - 5.0) * (k - 2.3) / 3.0 + rnorm();
    
    printf("%0.2f\t\t", Y[i]);
    for(j = 0; j < p; j++){
      printf("%0.2f\t", X[I(i, j, n)]);
    } 
    printf("\n");
  }
  printf("\n");
  
  hipMemcpy(dX, X, n * p * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(dY, Y, n * sizeof(float), hipMemcpyHostToDevice);

  hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, p, p, n, 
    a, dX, n, dX, n, b, dXtX, p);

  hipMemcpy(XtX, dXtX, p * p * sizeof(float), hipMemcpyDeviceToHost);

  printf("XtX\n");
  for(i = 0; i < p; i++){
    for(j = 0; j < p; j++){
      printf("%0.2f\t", XtX[I(i, j, p)]);
    }
    printf("\n");
  }
  printf("\n");

  culaInitialize();
  
  culaDeviceSgetrf(p, p, dXtX, p, ipiv);
  culaDeviceSgetri(p, dXtX, p, ipiv);
  
  hipMemcpy(XtX, dXtX, p * p * sizeof(float), hipMemcpyDeviceToHost);

  printf("XtX^(-1)\n");
  for(i = 0; i < p; i++){
    for(j = 0; j < p; j++){
      printf("%0.2f\t", XtX[I(i, j, p)]);
    }
    printf("\n");
  }
  printf("\n");
  
  hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, p, 1, n, 
    a, dX, n, dY, n, b, dXtY, p);

  hipblasSgemv(handle, HIPBLAS_OP_N, p, p, 
    a, dXtX, p, dXtY, 1, b, dbeta, 1);

  hipMemcpy(beta, dbeta, p * sizeof(float), hipMemcpyDeviceToHost);

  printf("CUBLAS/CULA matrix algebra parameter estimates:\n");
  for(i = 0; i < p; i++){
    printf("beta_%i = %0.2f\n", i, beta[i]);
  }
  printf("\n");


  culaSgels('N', n, p, 1, X, n, Y, n);

  printf("culaSgels Parameter estimates:\n");
  for(i = 0; i < p; i++){
    printf("beta_%i = %0.2f\n", i, Y[i]);
  }
  printf("\n");
  

  culaShutdown();
  hipblasDestroy(handle);

  free(a);
  free(b);
  free(X);
  free(XtX);
  free(XtY);
  free(beta);
  free(Y);
  hipFree(dX);
  hipFree(dXtX);
  hipFree(dXtY);
  hipFree(dbeta);
  hipFree(dY);
}