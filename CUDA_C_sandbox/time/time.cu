#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>


int main(){
  float   elapsedTime;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord( start, 0 );

  // SOME GPU WORK YOU WANT TIMED HERE

  hipEventRecord( stop, 0 );
  hipEventSynchronize( stop );
  hipEventElapsedTime( &elapsedTime, start, stop );
  hipEventDestroy( start );
  hipEventDestroy( stop );
  printf("GPU Time elapsed: %f milliseconds\n", elapsedTime);
}