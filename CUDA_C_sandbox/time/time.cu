#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>


int main(){
  float   elapsedTime;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord( start, 0 );

  // SOME GPU KERNEL YOU WANT TIMED HERE

  hipEventRecord( stop, 0 );
  hipEventSynchronize( stop );
  hipEventElapsedTime( &elapsedTime, start, stop );
  hipEventDestroy( start );
  hipEventDestroy( stop );
  printf("GPU Time elapsed: %f\n", elapsedTime);
}