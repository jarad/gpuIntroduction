#include <stdio.h> 
#include <stdlib.h> 
#include <math.h>
#include <time.h>
#include <unistd.h>
#include <hip/hip_runtime.h>
 

/*
 * This program computes the sum of the elements of 
 * vector v using the pairwise (cascading) sum algorithm.
 */

#define N 1024 // length of vector v. MUST BE A POWER OF 2.

// Fill the vector v with n random floating point numbers.
void vfill(float* v, int n){
  int i;
  for(i = 0; i < n; i++){
    v[i] = (float) rand() / RAND_MAX;
  }
}

// Print the vector v.
void vprint(float* v, int n){
  int i;
  printf("v = \n");
  for(i = 0; i < n; i++){
    printf("%7.3f\n", v[i]);
  }
  printf("\n");
}

// Pairwise-sum the elements of vector v and store the result in v[0]. 
__global__ void psum(float *v){ 
  int t = threadIdx.x; // Thread index.
  int n = blockDim.x; // Should be half the length of v.

  while (n != 0) {
    if(t < n)
      v[t] += v[t + n];  
    __syncthreads();    
    n /= 2; 
  }
}

// Linear sum the elements of vector v and return the result
float lsum(float *v, int len){
  float s = 0;
  int i;
  for(i = 0; i < len; i++){
    s += v[i];
  }
  return s;
}


int main (void){ 
  if(N % 2){
    printf("\nERROR: N is not a power of 2. Exiting.\n");
    exit(1);
  }

  float *v_h, *v_d; // host and device copies of our vector, respectively
  
  // dynamically allocate memory on the host for v_h
  v_h = (float*) malloc(N * sizeof(*v_h)); 
  
  // dynamically allocate memory on the device for v_d
  hipMalloc ((float**) &v_d, N *sizeof(*v_d)); 
  
  // Fill v_h with N random floating point numbers.
  vfill(v_h, N);
  
  // Print v_h to the console
  // vprint(v_h, N);
  
  // Write the contents of v_h to v_d
  hipMemcpy( v_d, v_h, N * sizeof(float), hipMemcpyHostToDevice );
    
  // compute the linear sum of the elements of v_h on the CPU and return the result
  // also, time the result.
  clock_t start = clock();
  float s = lsum(v_h, N);
  
  float elapsedTime = ((float) clock() - start) / CLOCKS_PER_SEC;
  printf("Linear Sum = %7.3f, CPU Time elapsed: %f\n", s, elapsedTime);
 
  // Compute the pairwise sum of the elements of v_d and store the result in v_d[0].
  // Also, time the computation.
  
  float   gpuElapsedTime;
  hipEvent_t gpuStart, gpuStop;
  hipEventCreate(&gpuStart);
  hipEventCreate(&gpuStop);
  hipEventRecord( gpuStart, 0 );

  psum<<< 1, N/2 >>>(v_d);
  
  hipEventRecord( gpuStop, 0 );
  hipEventSynchronize( gpuStop );
  hipEventElapsedTime( &gpuElapsedTime, gpuStart, gpuStop );
  hipEventDestroy( gpuStart );
  hipEventDestroy( gpuStop );
  
  // Write the pairwise sum, v_d[0], to v_h[0].
  hipMemcpy(v_h, v_d, sizeof(float), hipMemcpyDeviceToHost );
  
  // Print the pairwise sum.
  printf("Pairwise Sum = %7.3f, GPU Time elapsed: %f\n", v_h[0], gpuElapsedTime);
   
  // Free dynamically-allocated host memory
  free(v_h);

  // Free dynamically-allocated device memory    
  hipFree(&v_d);
}