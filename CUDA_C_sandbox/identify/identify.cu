
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void isExecuted(int *a_d, int blockid, int threadid){
  if(blockIdx.x == blockid && threadIdx.x == threadid)
    *a_d = 1;
}

int main(){

  int a = 0, *a_d;
  
  hipMalloc((void**) &a_d, sizeof(int));
  hipMemcpy(a_d, &a, sizeof(int), hipMemcpyHostToDevice);

  isExecuted<<<100,100>>>(a_d, 2, 4); 
  
  hipMemcpy(&a, a_d, sizeof(int), hipMemcpyDeviceToHost);

  printf("a = %d\n", a);
  hipFree(a_d);

}