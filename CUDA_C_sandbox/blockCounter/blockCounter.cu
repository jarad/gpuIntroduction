#include "hip/hip_runtime.h"
#include "../common/lock.h"
#define NBLOCKS_TRUE 512
#define NTHREADS_TRUE 512 * 2

__global__ void blockCounterUnlocked( int *nblocks ){
   if(threadIdx.x == 0){
    *nblocks = *nblocks + 1;
  }
}

__global__ void blockCounter1( Lock lock, int *nblocks ){
  if(threadIdx.x == 0){
    lock.lock();
    *nblocks = *nblocks + 1;
    lock.unlock();
  }
}

// THIS KERNEL WILL CREATE A DIVERGENCE CONDITION
// AND STALL OUT. DON'T USE IT.
__global__ void blockCounter2( Lock lock, int *nblocks ){
  lock.lock();
  if(threadIdx.x == 0){
    *nblocks = *nblocks + 1 ;
  }
  lock.unlock();
}


int main(){
  int nblocks_host, *nblocks_dev;
  Lock lock;
  float elapsedTime;
  hipEvent_t start, stop;
 
  hipMalloc((void**) &nblocks_dev, sizeof(int));
  

  //blockCounterUnlocked:

  nblocks_host = 0;
  hipMemcpy( nblocks_dev, &nblocks_host, sizeof(int), hipMemcpyHostToDevice );
  
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord( start, 0 );
  
  blockCounterUnlocked<<<NBLOCKS_TRUE, NTHREADS_TRUE>>>(nblocks_dev);

  hipEventRecord( stop, 0 );
  hipEventSynchronize( stop );
  hipEventElapsedTime( &elapsedTime, start, stop );

  hipEventDestroy( start );
  hipEventDestroy( stop ); 

  hipMemcpy( &nblocks_host, nblocks_dev, sizeof(int), hipMemcpyDeviceToHost );
  printf("blockCounterUnlocked <<< %d, %d >>> () counted %d blocks in %f ms.\n", 
        NBLOCKS_TRUE,
        NTHREADS_TRUE,
        nblocks_host,
        elapsedTime);
        
        
  //blockCounter1:

  nblocks_host = 0;
  hipMemcpy( nblocks_dev, &nblocks_host, sizeof(int), hipMemcpyHostToDevice );
  
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord( start, 0 );
  
  blockCounter1<<<NBLOCKS_TRUE, NTHREADS_TRUE>>>(lock, nblocks_dev);

  hipEventRecord( stop, 0 );
  hipEventSynchronize( stop );
  hipEventElapsedTime( &elapsedTime, start, stop );

  hipEventDestroy( start );
  hipEventDestroy( stop ); 

  hipMemcpy( &nblocks_host, nblocks_dev, sizeof(int), hipMemcpyDeviceToHost );
  printf("blockCounter1 <<< %d, %d >>> () counted %d blocks in %f ms.\n", 
        NBLOCKS_TRUE,
        NTHREADS_TRUE,
        nblocks_host,
        elapsedTime);      
                   
  hipFree(nblocks_dev); 
}