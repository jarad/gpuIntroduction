
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void colonel(int *dev_a){
  *dev_a = 1;
}

int main(){

  // Declare variables and allocate memory on the GPU.
  int a[1], *dev_a;
  hipMalloc((void**) &dev_a, sizeof(int));

  // Execute kernel and copy the result to CPU memory.
  colonel<<<1,1>>>(dev_a);
  hipMemcpy(a, dev_a, sizeof(int), hipMemcpyDeviceToHost);

  // Print result and free dynamically allocated memory.
  printf("a[0] = %d\n", a[0]); // REMEMBER: INDEXING IN C STARTS FROM 0.
  hipFree(dev_a);

}