#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
 

__global__ void colonel(int *a_d){
  *a_d = 2;
}

int main(){

  int a = 0, *a_d;
  
  hipMalloc((void**) &a_d, sizeof(int));
  hipMemcpy(a_d, &a, sizeof(int), hipMemcpyHostToDevice);

  colonel<<<100,100>>>(a_d); 
  
  hipMemcpy(&a, a_d, sizeof(int), hipMemcpyDeviceToHost);

  printf("a = %d\n", a);
  hipFree(a_d);

}