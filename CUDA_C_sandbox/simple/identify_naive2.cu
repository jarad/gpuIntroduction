
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void isExecuted(int *dev_a, int blockid, int threadid){
  
  if(blockIdx.x == blockid && threadIdx.x == threadid)
    *dev_a = 1;
  else
    *dev_a = 0;

}

int main(){

  // Declare variables and allocate memory on the GPU.
  int a[1], *dev_a;
  hipMalloc((void**) &dev_a, sizeof(int));

  // Execute kernel and copy the result to CPU memory.
  isExecuted<<<100,100>>>(dev_a, 2, 4); // NOTE: INDEXING OF THREADS AND BLOCKS STARTS FROM 0.
  hipMemcpy(a, dev_a, sizeof(int), hipMemcpyDeviceToHost);

  // Print result and free dynamically allocated memory.
  printf("a[0] = %d\n", a[0]); // REMEMBER: INDEXING IN C STARTS FROM 0.
  hipFree(dev_a);

}