/*
 * Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and 
 * international Copyright laws.  
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.   
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE 
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.  This source code is a "commercial item" as 
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of 
 * "commercial computer software" and "commercial computer software 
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995) 
 * and is provided to the U.S. Government only as a commercial end item.  
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

#ifndef _SCAN_BEST_KERNEL_H_
#define _SCAN_BEST_KERNEL_H_

#define NUM_BANKS 16
#define LOG_NUM_BANKS 4

// Define this to more rigorously avoid bank conflicts, even at the lower (root) levels of the tree
//#define ZERO_BANK_CONFLICTS 

#ifdef ZERO_BANK_CONFLICTS
#define CONFLICT_FREE_OFFSET(index) ((index) >> LOG_NUM_BANKS + (index) >> (2 * LOG_NUM_BANKS))
#else
#define CONFLICT_FREE_OFFSET(index) ((index) >> LOG_NUM_BANKS)
#endif

#ifdef CHECK_BANK_CONFLICTS
#define TEMP(index)   CUT_BANK_CHECKER(temp, index)
#else
#define TEMP(index)   temp[index]
#endif

///////////////////////////////////////////////////////////////////////////////
// Work-efficient compute implementation of scan, one thread per 2 elements
// Work-efficient: O(log(n)) steps, and O(n) adds.
// Also shared storage efficient: Uses n + n/NUM_BANKS shared memory -- no ping-ponging
// Also avoids most bank conflicts using single-element offsets every NUM_BANKS elements.
//
// In addition, If ZERO_BANK_CONFLICTS is defined, uses 
//     n + n/NUM_BANKS + n/(NUM_BANKS*NUM_BANKS) 
// shared memory. If ZERO_BANK_CONFLICTS is defined, avoids ALL bank conflicts using 
// single-element offsets every NUM_BANKS elements, plus additional single-element offsets 
// after every NUM_BANKS^2 elements.
//
// Uses a balanced tree type algorithm.  See Blelloch, 1990 "Prefix Sums 
// and Their Applications", or Prins and Chatterjee PRAM course notes:
// http://www.cs.unc.edu/~prins/Classes/203/Handouts/pram.pdf
// 
// This work-efficient version is based on the algorithm presented in Guy Blelloch's
// Excellent paper "Prefix sums and their applications".
// http://www-2.cs.cmu.edu/afs/cs.cmu.edu/project/scandal/public/papers/CMU-CS-90-190.html
//
// Pro: Work Efficient, very few bank conflicts (or zero if ZERO_BANK_CONFLICTS is defined)
// Con: More instructions to compute bank-conflict-free shared memory addressing,
// and slightly more shared memory storage used.
//
// @param g_odata  output data in global memory
// @param g_idata  input data in global memory
// @param n        input number of elements to scan from input data
__global__ void scan_best(float *g_odata, float *g_idata, int n)
{
    // Dynamically allocated shared memory for scan kernels
    extern  __shared__  float temp[];

    int thid = threadIdx.x;

    int ai = thid;
    int bi = thid + (n/2);

    // compute spacing to avoid bank conflicts
    int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
    int bankOffsetB = CONFLICT_FREE_OFFSET(bi);

    // Cache the computational window in shared memory
    TEMP(ai + bankOffsetA) = g_idata[ai]; 
    TEMP(bi + bankOffsetB) = g_idata[bi]; 

    int offset = 1;

    // build the sum in place up the tree
    for (int d = n/2; d > 0; d >>= 1)
    {
        __syncthreads();

        if (thid < d)      
        {
            int ai = offset*(2*thid+1)-1;
            int bi = offset*(2*thid+2)-1;

            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);

            TEMP(bi) += TEMP(ai);
        }

        offset *= 2;
    }

    // scan back down the tree

    // clear the last element
    if (thid == 0)
    {
        int index = n - 1;
        index += CONFLICT_FREE_OFFSET(index);
        TEMP(index) = 0;
    }   

    // traverse down the tree building the scan in place
    for (int d = 1; d < n; d *= 2)
    {
        offset /= 2;

        __syncthreads();

        if (thid < d)
        {
            int ai = offset*(2*thid+1)-1;
            int bi = offset*(2*thid+2)-1;

            ai += CONFLICT_FREE_OFFSET(ai);
            bi += CONFLICT_FREE_OFFSET(bi);

            float t  = TEMP(ai);
            TEMP(ai) = TEMP(bi);
            TEMP(bi) += t;
        }
    }

    __syncthreads();

    // write results to global memory
    g_odata[ai] = TEMP(ai + bankOffsetA); 
    g_odata[bi] = TEMP(bi + bankOffsetB); 
}
#endif // #ifndef _SCAN_BEST_KERNEL_H_

